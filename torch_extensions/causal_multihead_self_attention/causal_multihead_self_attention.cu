#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>
#include <cmath>
#include <algorithm>

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

#include <pybind11/pybind11.h>


#define ALL_THREADS_IN_WARP_MASK 0xffffffffu
#define THREADS_PER_WARP 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

namespace causal_multihead_self_attention {

// Taken from https://docs.pytorch.org/tutorials/advanced/cpp_custom_ops.html#setting-up-hybrid-python-c-registration,
// tweaked with https://stackoverflow.com/a/76669141.
PYBIND11_MODULE(causal_multihead_self_attention, m) {}

__device__ static inline float onlineSoftmaxSum(float const maxA,
                                                float const sumA,
                                                float const maxB,
                                                float const sumB) {
    if (sumA == 0.0f) {
        return sumB;
    } else if (sumB == 0.0f) {
        return sumA;
    } else if (maxA > maxB) {
        return sumB * expf(maxB - maxA) + sumA;
    } else {
        return sumB + sumA * expf(maxA - maxB);
    }
}

template <int d_head, int d_model, int B_c, int B_r>
__global__ void causal_multihead_self_attention_kernel(float const* const Q_HBM,  // size Nxd_model
                                                       float const* const K_HBM,  // size Nxd_model
                                                       float const* const V_HBM,  // size Nxd_model
                                                       float* const O_HBM,        // size Nxd_model
                                                       int const N) {
    extern __shared__ float sharedMemory[];
    int const T_c = CEIL_DIV(N, B_c);
    float const temperature = sqrt(d_head);

    int const B_r_bounds_checked_for_last_row = min(B_r, N - blockIdx.x * B_r);
    int const d_min_for_head = blockIdx.y * d_head;
    int const Q_row_length = d_head;
    int const O_row_length = d_head;
    // For alleviating shared memory bank conflicts
    int const K_row_length = d_head + 4;
    int const V_row_length = B_c + 4;

    float* const Q = sharedMemory;
    float* const K = Q + B_r * Q_row_length;
    float* const V = K + B_c * K_row_length;
    float* const S = V + d_head * V_row_length;
    float* const O = S + B_c * B_r;
    float4* const Q_float4 = reinterpret_cast<float4*>(Q);
    float4* const K_float4 = reinterpret_cast<float4*>(K);
    float4* const V_float4 = reinterpret_cast<float4*>(V);
    float4* const S_float4 = reinterpret_cast<float4*>(S);
    float4* const O_float4 = reinterpret_cast<float4*>(O);
    float4 const* const Q_HBM_float4 = reinterpret_cast<float4 const*>(Q_HBM);
    float4 const* const K_HBM_float4 = reinterpret_cast<float4 const*>(K_HBM);
    float4 const* const V_HBM_float4 = reinterpret_cast<float4 const*>(V_HBM);
    float4* const O_HBM_float4 = reinterpret_cast<float4*>(O_HBM);

    float4 const zero_float4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    // Load Q, using threadIdx.x to help along the d_head dimension (for memory coalescing) and
    // threadIdx.y to help along the B_r dimension.
    for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
        for (int B_r_index = threadIdx.y; B_r_index < B_r_bounds_checked_for_last_row; B_r_index += blockDim.y) {
            int const row_index = blockIdx.x * B_r + B_r_index;
            Q_float4[B_r_index * (Q_row_length/4) + d_index] = Q_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
            O_float4[B_r_index * (O_row_length/4) + d_index] = zero_float4;
        }
    }

    float S_row_old_global_sum = 0.0f;
    float S_row_old_global_max = -INFINITY;

    // Iterate horizontally through different S blocks.
    for (int T_c_index = 0; T_c_index < T_c; T_c_index++) {
        int const num_cols_beyond_this_block_start = N - T_c_index * B_c;
        int const B_c_bounds_checked_for_last_column = min(B_c, num_cols_beyond_this_block_start);
        // Load K using threadIdx.x to help along the d_head dimension (for memory coalescing) and
        // threadIdx.y to help along the B_c dimension.
        for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
            for (int B_c_index = threadIdx.y; B_c_index < B_c_bounds_checked_for_last_column; B_c_index += blockDim.y) {
                int const row_index = T_c_index * B_c + B_c_index;
                K_float4[B_c_index * (K_row_length / 4) + d_index] = K_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
            }
        }

        // Make sure we're done writing Q, K, and V before we read them.
        __syncthreads();

        int const B_r_index = threadIdx.y;
        int const top_row_absolute = B_r * blockIdx.x;
        int const bottow_row_absolute = top_row_absolute + B_r - 1;
        int const left_column_absolute = T_c_index * B_c;

        if (left_column_absolute > bottow_row_absolute) {
            // This entire block is masked out by causal masking.
            goto write_output;
        }

        bool const row_in_bounds = B_r_index < B_r_bounds_checked_for_last_row;
        int const row_absolute = top_row_absolute + B_r_index;
        int const column_upper_bound_absolute = row_absolute + 1;
        int const column_upper_bound_within_tile = column_upper_bound_absolute - left_column_absolute;
        int const column_upper_bound = min(column_upper_bound_within_tile, B_c_bounds_checked_for_last_column);
        bool const start_column_in_row_unmasked = column_upper_bound > 0;
        bool const col_unmasked = threadIdx.x < column_upper_bound;
        float S_row_new_global_sum;
        float S_row_new_global_max;
        float S_val_for_thread = 0.0f;
        if (col_unmasked && row_in_bounds) {
            // Compute S.
            #pragma unroll
            for (int d_index = 0; d_index < d_head / 4; d_index++) {
                float4 const Q_val_float4 = Q_float4[B_r_index * (Q_row_length / 4) + d_index];
                float4 const K_val_float4 = K_float4[threadIdx.x * (K_row_length / 4) + d_index];
                S_val_for_thread += Q_val_float4.w * K_val_float4.w;
                S_val_for_thread += Q_val_float4.x * K_val_float4.x;
                S_val_for_thread += Q_val_float4.y * K_val_float4.y;
                S_val_for_thread += Q_val_float4.z * K_val_float4.z;
            }
            S_val_for_thread = S_val_for_thread / temperature;
            S[B_r_index * B_c + threadIdx.x] = S_val_for_thread;
        }

        float localSum;
        float localMax;
        if (row_in_bounds && start_column_in_row_unmasked) {
            // Gather the values for localSum and localMax on threadIdx.x == 0.
            // ASSUMPTION: blockDim.x == 32
            localSum = col_unmasked ? 1.0f : 0.0f;
            localMax = col_unmasked ? S_val_for_thread : -INFINITY;
            for (int numActiveThreads = THREADS_PER_WARP / 2; numActiveThreads >= 1; numActiveThreads /= 2) {
                float const incomingSum = __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
                float const incomingMax = __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreads);
                localSum = onlineSoftmaxSum(localMax, localSum, incomingMax, incomingSum);
                localMax = max(localMax, incomingMax);
            }
        }

        // Load V.
        for (int d_index = threadIdx.y; d_index < d_head / 4; d_index += blockDim.y) {
            for (int B_c_index = threadIdx.x; B_c_index < B_c_bounds_checked_for_last_column; B_c_index += blockDim.x) {
                int const row_index = T_c_index * B_c + B_c_index;
                float4 const V_val_float4 = V_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
                // Store V transposed so we can read with float4 later.
                V[(4 * d_index + 0) * V_row_length + B_c_index] = V_val_float4.x;
                V[(4 * d_index + 1) * V_row_length + B_c_index] = V_val_float4.y;
                V[(4 * d_index + 2) * V_row_length + B_c_index] = V_val_float4.z;
                V[(4 * d_index + 3) * V_row_length + B_c_index] = V_val_float4.w;
            }
        }

        if (row_in_bounds && start_column_in_row_unmasked) {
            // Broadcast the values for localSum and localMax from threadIdx.x == 0 to the other threads in the warp.
            localSum = __shfl_sync(ALL_THREADS_IN_WARP_MASK, localSum, 0);
            localMax = __shfl_sync(ALL_THREADS_IN_WARP_MASK, localMax, 0);

            S_row_new_global_sum = onlineSoftmaxSum(localMax, localSum, S_row_old_global_max, S_row_old_global_sum);
            S_row_new_global_max = max(localMax, S_row_old_global_max);
        }

        // Make sure we're done writing S before we read it.
        __syncthreads();

        if (row_in_bounds && start_column_in_row_unmasked) {
            // Compute P and O
            for (int d_index = threadIdx.x; d_index < d_head; d_index += blockDim.x) {
                float PV_val = 0.0f;
                int V_B_c_index = 0;
                for (; V_B_c_index < (column_upper_bound / 4) * 4; V_B_c_index += 4) {
                    float4 const S_val_float4 = S_float4[B_r_index * (B_c / 4) + (V_B_c_index / 4)];
                    float4 const V_val_float4 = V_float4[d_index * (V_row_length / 4) + (V_B_c_index / 4)];
                    PV_val += expf(S_val_float4.x - S_row_new_global_max) * V_val_float4.x;
                    PV_val += expf(S_val_float4.y - S_row_new_global_max) * V_val_float4.y;
                    PV_val += expf(S_val_float4.z - S_row_new_global_max) * V_val_float4.z;
                    PV_val += expf(S_val_float4.w - S_row_new_global_max) * V_val_float4.w;
                }
                for (; V_B_c_index < column_upper_bound; V_B_c_index += 1) {
                    float const S_val = S[B_r_index * B_c + V_B_c_index];
                    PV_val += expf(S_val - S_row_new_global_max) * V[d_index * V_row_length + V_B_c_index];
                }
                int const OIndexForThread = B_r_index * O_row_length + d_index;
                O[OIndexForThread] = (O[OIndexForThread] * expf(S_row_old_global_max - S_row_new_global_max) * S_row_old_global_sum + PV_val) / S_row_new_global_sum;
            }
        }

        S_row_old_global_sum = S_row_new_global_sum;
        S_row_old_global_max = S_row_new_global_max;

        // Make sure we're done reading S, Q, K, and V before we write them, and done writing O before we read it.
        __syncthreads();
    }

    // Write O_HBM
write_output:
    for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
        for (int B_r_index = threadIdx.y; B_r_index < B_r_bounds_checked_for_last_row; B_r_index += blockDim.y) {
            int const row_index = blockIdx.x * B_r + B_r_index;
            O_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index] = O_float4[B_r_index * (O_row_length/4) + d_index];
        }
    }
}


// Q, K, V, output are device pointers
void causal_multihead_self_attention(float const* const Q,  // size Nxd
                                     float const* const K,  // size Nxd
                                     float const* const V,  // size Nxd
                                     float* const output,   // size Nxd
                                     int const N,
                                     int const d_model,
                                     int const num_heads) {
    int maxSharedMemory;
    gpuErrchk(hipDeviceGetAttribute(&maxSharedMemory, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));

    int const d_head = d_model / num_heads;

    int constexpr B_c = 32;
    int constexpr B_r = 32;
    int const T_r = CEIL_DIV(N, B_r);

    dim3 const blocksPerGrid(T_r, num_heads);
    dim3 const threadsPerBlock(B_c, B_r);
    int const sharedMemoryBytes = (B_r * d_head          // Q
                                   + B_c * (d_head + 4)  // K
                                   + (B_c + 4) * d_head  // V
                                   + B_r * B_c           // S
                                   + B_r * d_head)       // O
                                  * sizeof(float);
    if (d_head != 64) {
        throw std::invalid_argument("Head dimension must be 64.");
    }
    if (d_model == 768) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 768, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 768, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1024) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1024, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1024, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1280) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1280, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1280, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1600) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1600, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1600, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else {
        throw std::invalid_argument("Model dimension must be 768, 1024, 1280, or 1600.");
    }
    gpuErrchk(hipPeekAtLastError());

#ifdef DEBUG
    std::cout << "T_r: " << T_r << std::endl;
    std::cout << "num_heads: " << num_heads << std::endl;
    std::cout << "B_c: " << B_c << std::endl;
    std::cout << "B_r: " << B_r << std::endl;
    std::cout << "N: " << N << std::endl;
    std::cout << "d_model: " << d_model << std::endl;
    std::cout << "d_head: " << d_head << std::endl;
    std::cout << "num_heads: " << num_heads << std::endl;
#endif
}

torch::Tensor causal_multihead_self_attention_torch(torch::Tensor Q,
                                                    torch::Tensor K,
                                                    torch::Tensor V,
                                                    int64_t num_heads) {
    TORCH_CHECK(Q.is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K.is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V.is_cuda(), "V must be a CUDA tensor");

    TORCH_CHECK(Q.dtype() == torch::kFloat32, "Q must be float32");
    TORCH_CHECK(K.dtype() == torch::kFloat32, "K must be float32");
    TORCH_CHECK(V.dtype() == torch::kFloat32, "V must be float32");

    TORCH_CHECK(Q.dim() == 2, "Q must be a 2D tensor");
    TORCH_CHECK(K.dim() == 2, "K must be a 2D tensor");
    TORCH_CHECK(V.dim() == 2, "V must be a 2D tensor");

    TORCH_CHECK(Q.is_contiguous(), "Q must be contiguous")
    TORCH_CHECK(K.is_contiguous(), "K must be contiguous")
    TORCH_CHECK(V.is_contiguous(), "V must be contiguous")

    int N = Q.size(0);
    int d = Q.size(1);

    TORCH_CHECK(K.size(0) == N, "K must have the same sequence length as Q");
    TORCH_CHECK(V.size(0) == N, "V must have the same sequence length as Q");
    TORCH_CHECK(K.size(1) == d, "K must have the same feature dimension as Q");
    TORCH_CHECK(V.size(1) == d, "V must have the same feature dimension as Q");

    TORCH_CHECK(d % num_heads == 0, "Feature dimension of Q must be evenly divisible by the number of heads");

    torch::Tensor output = torch::empty({N, d}, Q.options());

    // Call the kernel launcher
    causal_multihead_self_attention(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        output.data_ptr<float>(),
        N, d, (int)num_heads
    );

    return output;
}

TORCH_LIBRARY(causal_multihead_self_attention, m) {
   // Note that "float" in the schema corresponds to the C++ double type
   // and the Python float type.
   m.def("causal_multihead_self_attention_torch(Tensor Q, Tensor K, Tensor V, int num_heads) -> Tensor");
 }

TORCH_LIBRARY_IMPL(causal_multihead_self_attention, CUDA, m) {
  m.impl("causal_multihead_self_attention_torch", &causal_multihead_self_attention_torch);
}

}
