#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>
#include <cmath>
#include <algorithm>

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

#include <pybind11/pybind11.h>


#define THREADS_PER_WARP 32

int constexpr NUM_COLS_PER_THREAD = 2;
unsigned int constexpr ALL_THREADS_IN_WARP_MASK = 0xffffffffu;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define CEIL_DIV(dividend, divisor) (((dividend) + (divisor) - 1) / (divisor))

namespace causal_multihead_self_attention {

// Taken from https://docs.pytorch.org/tutorials/advanced/cpp_custom_ops.html#setting-up-hybrid-python-c-registration,
// tweaked with https://stackoverflow.com/a/76669141.
PYBIND11_MODULE(causal_multihead_self_attention, m) {}

__device__ static inline float onlineSoftmaxSum(float const maxA,
                                                float const sumA,
                                                float const maxB,
                                                float const sumB) {
    if (sumA == 0.0f) {
        return sumB;
    } else if (sumB == 0.0f) {
        return sumA;
    } else if (maxA > maxB) {
        return sumB * expf(maxB - maxA) + sumA;
    } else {
        return sumB + sumA * expf(maxA - maxB);
    }
}

template <int d_head, int d_model, int B_c, int B_r>
__global__ void causal_multihead_self_attention_kernel(float const* const __restrict__ Q_HBM,  // size Nxd_model
                                                       float const* const __restrict__ K_HBM,  // size Nxd_model
                                                       float const* const __restrict__ V_HBM,  // size Nxd_model
                                                       float* const __restrict__ O_HBM,        // size Nxd_model
                                                       int const N) {
    extern __shared__ float sharedMemory[];
    int const T_c = CEIL_DIV(N, B_c);
    float const temperature = sqrt(d_head);

    int const B_r_bounds_checked_for_last_row = min(B_r, N - blockIdx.x * B_r);
    int const d_min_for_head = blockIdx.y * d_head;
    int const Q_row_length = d_head;
    int const O_row_length = d_head;
    // For alleviating shared memory bank conflicts
    int const K_row_length = d_head + 4;

    float* const Q = sharedMemory;
    float* const K = Q + B_r * Q_row_length;
    float* const V = K + B_c * K_row_length;
    float* const S = V + B_c * d_head;
    float* const O = S + B_c * B_r;
    float4* const Q_float4 = reinterpret_cast<float4*>(Q);
    float4* const K_float4 = reinterpret_cast<float4*>(K);
    float4* const V_float4 = reinterpret_cast<float4*>(V);
    float4* const S_float4 = reinterpret_cast<float4*>(S);
    float4* const O_float4 = reinterpret_cast<float4*>(O);
    float4 const* const Q_HBM_float4 = reinterpret_cast<float4 const*>(Q_HBM);
    float4 const* const K_HBM_float4 = reinterpret_cast<float4 const*>(K_HBM);
    float4 const* const V_HBM_float4 = reinterpret_cast<float4 const*>(V_HBM);
    float4* const O_HBM_float4 = reinterpret_cast<float4*>(O_HBM);

    float4 const zero_float4 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    // Load Q, using threadIdx.x to help along the d_head dimension (for memory coalescing) and
    // threadIdx.y to help along the B_r dimension.
    for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
        for (int B_r_index = threadIdx.y; B_r_index < B_r_bounds_checked_for_last_row; B_r_index += blockDim.y) {
            int const row_index = blockIdx.x * B_r + B_r_index;
            Q_float4[B_r_index * (Q_row_length/4) + d_index] = Q_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
            O_float4[B_r_index * (O_row_length/4) + d_index] = zero_float4;
        }
    }

    float S_row_old_global_sum = 0.0f;
    float S_row_old_global_max = -INFINITY;

    // Iterate horizontally through different S blocks.
    for (int T_c_index = 0; T_c_index < T_c; T_c_index++) {
        int const top_row_absolute = B_r * blockIdx.x;
        int const bottow_row_absolute = top_row_absolute + B_r - 1;
        int const left_column_absolute = T_c_index * B_c;

        if (left_column_absolute > bottow_row_absolute) {
            // This entire block is masked out by causal masking.
            break;
        }

        int const num_cols_beyond_this_block_start = N - T_c_index * B_c;
        int const B_c_bounds_checked_for_last_column = min(B_c, num_cols_beyond_this_block_start);
        // Load K and V using threadIdx.x to help along the d_head dimension (for memory coalescing) and
        // threadIdx.y to help along the B_c dimension.
        for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
            for (int B_c_index = threadIdx.y; B_c_index < B_c_bounds_checked_for_last_column; B_c_index += blockDim.y) {
                int const row_index = T_c_index * B_c + B_c_index;
                K_float4[B_c_index * (K_row_length / 4) + d_index] = K_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
                V_float4[B_c_index * (d_head / 4) + d_index] = V_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index];
            }
        }

        // Make sure we're done writing Q, K, and V before we read them.
        __syncthreads();

        // Iterate vertically within the S block.
        // Since we use __syncthreads in this loop we have to make sure threads don't exit the function early.
        for (int B_r_index = threadIdx.y; B_r_index < CEIL_DIV(B_r_bounds_checked_for_last_row, blockDim.y) * blockDim.y; B_r_index += blockDim.y) {
            bool const row_in_bounds = B_r_index < B_r_bounds_checked_for_last_row;
            int const row_absolute = top_row_absolute + B_r_index;
            int const column_upper_bound_absolute = row_absolute + 1;
            int const column_upper_bound_within_tile = column_upper_bound_absolute - left_column_absolute;
            int const column_upper_bound = min(column_upper_bound_within_tile, B_c_bounds_checked_for_last_column);
            bool const start_column_in_row_unmasked = column_upper_bound > 0;
            int const left_S_val_column = NUM_COLS_PER_THREAD * threadIdx.x;
            bool const left_S_val_unmasked = left_S_val_column < column_upper_bound;
            float S_row_new_global_sum;
            float S_row_new_global_max;
            float localSum = 0.0f;
            float localMax = -INFINITY;

            // Initialize S to zero.
            float S_registers[NUM_COLS_PER_THREAD];
            #pragma unroll
            for (int S_reg_col = 0; S_reg_col < NUM_COLS_PER_THREAD; S_reg_col++) {
                S_registers[S_reg_col] = 0.0f;
            }

            if (row_in_bounds && left_S_val_unmasked) {
                // Compute S.
                #pragma unroll
                for (int d_index = 0; d_index < d_head / 4; d_index++) {
                    float4 const Q_val_float4 = Q_float4[B_r_index * (Q_row_length / 4) + d_index];

                    #pragma unroll
                    for (int S_reg_col = 0; S_reg_col < NUM_COLS_PER_THREAD; S_reg_col++) {
                        int const S_val_column = left_S_val_column + S_reg_col;
                        float4 const K_val_float4 = K_float4[S_val_column * (K_row_length / 4) + d_index];
                        S_registers[S_reg_col] += Q_val_float4.w * K_val_float4.w;
                        S_registers[S_reg_col] += Q_val_float4.x * K_val_float4.x;
                        S_registers[S_reg_col] += Q_val_float4.y * K_val_float4.y;
                        S_registers[S_reg_col] += Q_val_float4.z * K_val_float4.z;
                    }
                }

                // Write S to shared memory and compute localSum and localMax.
                #pragma unroll
                for (int S_reg_col = 0; S_reg_col < NUM_COLS_PER_THREAD; S_reg_col++) {
                    int const S_val_column = left_S_val_column + S_reg_col;
                    bool const S_val_unmasked = S_val_column < column_upper_bound;
                    if (S_val_unmasked) {
                        S_registers[S_reg_col] = S_registers[S_reg_col] / temperature;
                        S[B_r_index * B_c + S_val_column] = S_registers[S_reg_col];

                        localSum = onlineSoftmaxSum(localMax, localSum, S_registers[S_reg_col], 1.0f);
                        localMax = max(localMax, S_registers[S_reg_col]);
                    }
                }
            }

            // Gather the values for localSum and localMax on threadIdx.x == 0.
            // Skip bound checks and causal masking checks because it's not as simple as checking 1 row since the
            // warp shuffle spans 2 rows. Checking could be done as an optimization, you just need to be careful
            // not to cause things to run forever when N is odd.
            // ASSUMPTION: blockDim.x == 16
            for (int numActiveThreads = THREADS_PER_WARP / 4; numActiveThreads >= 1; numActiveThreads /= 2) {
                float const incomingSum = __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localSum, numActiveThreads);
                float const incomingMax = __shfl_down_sync(ALL_THREADS_IN_WARP_MASK, localMax, numActiveThreads);
                localSum = onlineSoftmaxSum(localMax, localSum, incomingMax, incomingSum);
                localMax = max(localMax, incomingMax);
            }

            // Broadcast the values for localSum and localMax from threadIdx.x == 0 to the other threads in the half-warp..
            // See previous warp shuffle comment about skipping bounds checks.
            int const source_lane = threadIdx.y % 2 == 0 ? 0 : 16;
            localSum = __shfl_sync(ALL_THREADS_IN_WARP_MASK, localSum, source_lane);
            localMax = __shfl_sync(ALL_THREADS_IN_WARP_MASK, localMax, source_lane);

            S_row_new_global_sum = onlineSoftmaxSum(localMax, localSum, S_row_old_global_max, S_row_old_global_sum);
            S_row_new_global_max = max(localMax, S_row_old_global_max);

            // Make sure we're done writing S before we read it.
            __syncthreads();

            if (row_in_bounds && start_column_in_row_unmasked) {
                // Compute P and O
                for (int d_index = threadIdx.x * NUM_COLS_PER_THREAD; d_index < d_head; d_index += blockDim.x * NUM_COLS_PER_THREAD) {
                    float O_registers[NUM_COLS_PER_THREAD];
                    // Set O vals to 0.
                    #pragma unroll
                    for (int O_reg_col = 0; O_reg_col < NUM_COLS_PER_THREAD; O_reg_col++) {
                        O_registers[O_reg_col] = 0.0f;
                    }

                    // Compute O vals in "strides" of 4 elements at a time.
                    int V_B_c_index = 0;
                    for (; V_B_c_index < (column_upper_bound / 4) * 4; V_B_c_index += 4) {
                        float4 S_val_float4 = S_float4[B_r_index * (B_c / 4) + (V_B_c_index / 4)];
                        S_val_float4.x = expf(S_val_float4.x - S_row_new_global_max);
                        S_val_float4.y = expf(S_val_float4.y - S_row_new_global_max);
                        S_val_float4.z = expf(S_val_float4.z - S_row_new_global_max);
                        S_val_float4.w = expf(S_val_float4.w - S_row_new_global_max);

                        #pragma unroll
                        for (int O_reg_col = 0; O_reg_col < NUM_COLS_PER_THREAD; O_reg_col++) {
                            O_registers[O_reg_col] += S_val_float4.x * V[(V_B_c_index + 0) * d_head + d_index + O_reg_col];
                            O_registers[O_reg_col] += S_val_float4.y * V[(V_B_c_index + 1) * d_head + d_index + O_reg_col];
                            O_registers[O_reg_col] += S_val_float4.z * V[(V_B_c_index + 2) * d_head + d_index + O_reg_col];
                            O_registers[O_reg_col] += S_val_float4.w * V[(V_B_c_index + 3) * d_head + d_index + O_reg_col];
                        }
                    }

                    // Compute O vals in a "stride" of 1 element at a time.
                    for (; V_B_c_index < column_upper_bound; V_B_c_index += 1) {
                        float S_val = S[B_r_index * B_c + V_B_c_index];
                        S_val = expf(S_val - S_row_new_global_max);

                        #pragma unroll
                        for (int O_reg_col = 0; O_reg_col < NUM_COLS_PER_THREAD; O_reg_col++) {
                            O_registers[O_reg_col] += S_val * V[V_B_c_index * d_head + d_index + O_reg_col];
                        }
                    }

                    // Compute and write O values
                    #pragma unroll
                    for (int O_reg_col = 0; O_reg_col < NUM_COLS_PER_THREAD; O_reg_col++) {
                        int const OIndexForThread = B_r_index * O_row_length + d_index + O_reg_col;
                        O[OIndexForThread] = (O[OIndexForThread] * expf(S_row_old_global_max - S_row_new_global_max) * S_row_old_global_sum + O_registers[O_reg_col]) / S_row_new_global_sum;
                    }
                }
            }

            S_row_old_global_sum = S_row_new_global_sum;
            S_row_old_global_max = S_row_new_global_max;

            // Make sure we're done reading S, Q, K, and V before we write them, and done writing O before we read it.
            __syncthreads();
        }
    }

    // Write O_HBM
    for (int d_index = threadIdx.x; d_index < d_head / 4; d_index += blockDim.x) {
        for (int B_r_index = threadIdx.y; B_r_index < B_r_bounds_checked_for_last_row; B_r_index += blockDim.y) {
            int const row_index = blockIdx.x * B_r + B_r_index;
            O_HBM_float4[row_index * (d_model / 4) + (d_min_for_head / 4) + d_index] = O_float4[B_r_index * (O_row_length/4) + d_index];
        }
    }
}


// Q, K, V, output are device pointers
void causal_multihead_self_attention(float const* const Q,  // size Nxd
                                     float const* const K,  // size Nxd
                                     float const* const V,  // size Nxd
                                     float* const output,   // size Nxd
                                     int const N,
                                     int const d_model,
                                     int const num_heads) {
    // 64KB is available on Turing GPUs. Change this if using a GPU with a larger value.
    int maxSharedMemory = 65536;

    int const d_head = d_model / num_heads;

    int constexpr B_c = 32;
    int constexpr B_r = 32;
    int const T_r = CEIL_DIV(N, B_r);

    dim3 const blocksPerGrid(T_r, num_heads);
    dim3 const threadsPerBlock(16, B_r);
    int const sharedMemoryBytes = (B_r * d_head          // Q
                                   + B_c * (d_head + 4)  // K
                                   + B_c * d_head        // V
                                   + B_r * B_c           // S
                                   + B_r * d_head)       // O
                                  * sizeof(float);
    if (d_head != 64) {
        throw std::invalid_argument("Head dimension must be 64.");
    }
    if (d_model == 768) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 768, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 768, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1024) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1024, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1024, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1280) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1280, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1280, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else if (d_model == 1600) {
        gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(causal_multihead_self_attention_kernel<64), 1600, B_c, B_r>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));
        causal_multihead_self_attention_kernel<64, 1600, B_c, B_r><<<blocksPerGrid, threadsPerBlock, sharedMemoryBytes>>>(Q, K, V, output, N);
    } else {
        throw std::invalid_argument("Model dimension must be 768, 1024, 1280, or 1600.");
    }
    gpuErrchk(hipPeekAtLastError());

#ifdef DEBUG
    std::cout << "T_r: " << T_r << std::endl;
    std::cout << "num_heads: " << num_heads << std::endl;
    std::cout << "B_c: " << B_c << std::endl;
    std::cout << "B_r: " << B_r << std::endl;
    std::cout << "N: " << N << std::endl;
    std::cout << "d_model: " << d_model << std::endl;
    std::cout << "d_head: " << d_head << std::endl;
    std::cout << "num_heads: " << num_heads << std::endl;
#endif
}

torch::Tensor causal_multihead_self_attention_torch(torch::Tensor Q,
                                                    torch::Tensor K,
                                                    torch::Tensor V,
                                                    int64_t num_heads) {
    TORCH_CHECK(Q.is_cuda(), "Q must be a CUDA tensor");
    TORCH_CHECK(K.is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(V.is_cuda(), "V must be a CUDA tensor");

    TORCH_CHECK(Q.dtype() == torch::kFloat32, "Q must be float32");
    TORCH_CHECK(K.dtype() == torch::kFloat32, "K must be float32");
    TORCH_CHECK(V.dtype() == torch::kFloat32, "V must be float32");

    TORCH_CHECK(Q.dim() == 2, "Q must be a 2D tensor");
    TORCH_CHECK(K.dim() == 2, "K must be a 2D tensor");
    TORCH_CHECK(V.dim() == 2, "V must be a 2D tensor");

    TORCH_CHECK(Q.is_contiguous(), "Q must be contiguous")
    TORCH_CHECK(K.is_contiguous(), "K must be contiguous")
    TORCH_CHECK(V.is_contiguous(), "V must be contiguous")

    int N = Q.size(0);
    int d = Q.size(1);

    TORCH_CHECK(K.size(0) == N, "K must have the same sequence length as Q");
    TORCH_CHECK(V.size(0) == N, "V must have the same sequence length as Q");
    TORCH_CHECK(K.size(1) == d, "K must have the same feature dimension as Q");
    TORCH_CHECK(V.size(1) == d, "V must have the same feature dimension as Q");

    TORCH_CHECK(d % num_heads == 0, "Feature dimension of Q must be evenly divisible by the number of heads");

    torch::Tensor output = torch::empty({N, d}, Q.options());

    // Call the kernel launcher
    causal_multihead_self_attention(
        Q.data_ptr<float>(),
        K.data_ptr<float>(),
        V.data_ptr<float>(),
        output.data_ptr<float>(),
        N, d, (int)num_heads
    );

    return output;
}

TORCH_LIBRARY(causal_multihead_self_attention, m) {
   // Note that "float" in the schema corresponds to the C++ double type
   // and the Python float type.
   m.def("causal_multihead_self_attention_torch(Tensor Q, Tensor K, Tensor V, int num_heads) -> Tensor");
 }

TORCH_LIBRARY_IMPL(causal_multihead_self_attention, CUDA, m) {
  m.impl("causal_multihead_self_attention_torch", &causal_multihead_self_attention_torch);
}

}
